#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include<malloc.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include<cmath>

void matrix_mul_cpu(int n, float* a, float* b, float* c)
{
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
        {
            float s = 0.0;
            for (int k = 0; k < n; ++k)
                s += a[i * n + k] * b[k * n + j];               //A(i,k)*B(k,j)
            c[i * n + j] = s;
        }
}

__global__ void matrix_mul_gpu(int n, float* a, float* b, float* c)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    float s = 0.0;
    for (int k = 0; k < n; ++k)
        s += a[bid * n + k] * b[k * n + tid];               //A(i,k)*B(k,j)
    c[bid * n + tid] = s;
}
/*__global__ void transpose(int n, float* b, float* bt)
{
    bt[blockIdx.x * n + threadIdx.x] = b[threadIdx.x * n + blockIdx.x];
}
__global__ void matrix_mul_gpu2(int n, float* a, float* b, float* c)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    float s = 0.0;
    for (int k = 0; k < n; ++k)
        s += a[bid * n + k] * b[tid * n + k];          //A*BT  连续访问
    c[bid * n + tid] = s;
}*/
void check_data(int n, float* c, float* c1)
{
    float s = 0.0;
    for (int i = 0; i < n * n; ++i)
        s += fabs(c[i] - c1[i]);
    printf("Total error is %f \n", s);
}
int main()
{
    const int n = 1024, M = n * n * sizeof(float);
    float* a = (float*)malloc(M);
    float* b = (float*)malloc(M);
    float* c = (float*)malloc(M);
    float* c1 = (float*)malloc(M);
    float* d_a, * d_b, * d_c, * d_bT;

    for (int i = 0; i < n * n; ++i)
    {
        a[i] = (float)(rand() % 100);
        b[i] = (float)(rand() % 100);
    }

    clock_t time1, time2, time3, time4;
    time1 = clock();
    matrix_mul_cpu(n, a, b, c);
    time2 = clock();
    printf("Time for CPU run  is: %f seconds \n", (double)(time2 - time1) / CLOCKS_PER_SEC);

    hipMalloc(&d_a, M);
    hipMalloc(&d_b, M);
    hipMalloc(&d_c, M);
    hipMemcpy(d_a, a, M, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M, hipMemcpyHostToDevice);

    time3 = clock();
    matrix_mul_gpu << <n, n >> > (n, d_a, d_b, d_c);  // 计算C=A*B, 每个线程计算一个点
    hipDeviceSynchronize();
    hipMemcpy(c1, d_c, M, hipMemcpyDeviceToHost);
    time4 = clock();
    printf("Time for GPU run  is: %f seconds \n", (double)(time4 - time3) / CLOCKS_PER_SEC);
    check_data(n, c, c1);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    //hipFree(d_bT);// by using transpose 

    /*time1 = clock();
    hipMalloc(&d_a, M);
    hipMalloc(&d_b, M);
    hipMalloc(&d_c, M);
    hipMalloc(&d_bT, M);
    hipMemcpy(d_a, a, M, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M, hipMemcpyHostToDevice);

    transpose << <n, n >> > (n, d_b, d_bT);
    hipDeviceSynchronize();
    matrix_mul_gpu2 << <n, n >> > (n, d_a, d_bT, d_c);
    hipMemcpy(c1, d_c, M, hipMemcpyDeviceToHost);
    time2 = clock();
    printf("Time for GPU run  is: %f seconds \n", (double)(time2 - time1) / CLOCKS_PER_SEC);
    check_data(n, c, c1);

 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_bT);*/ 
    free(a);
    free(b);
    free(c);
    return 0;
}




