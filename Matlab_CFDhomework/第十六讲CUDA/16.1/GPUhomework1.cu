#include "hip/hip_runtime.h"
# include <math.h>
#include <time.h>
#include <stdio.h>
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
#include<malloc.h>
#include <stdlib.h>
void sum_cpu(const int m)//��cpu�����������1+2+...+5000000000,Ϊ������gpu���м��ٵ�Ч�����Ĵ�������
{
	double s = 0.0;
	for (int i = 0; i <m+1; i++)
	{
		s = s + i;
	}
	printf("s=%20.16e \n", s);
}
__global__ void sum_gpu(const int n, double* sum_local_gpu)
{
	double s = 0.0;
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (int j = i; j < n+1; j=j+gridDim.x*blockDim.x)
	{
		s = s + j;
	}
	sum_local_gpu[i] = s;
}
int main()
{
	const long end = 5000000000, p = 100, mp = 100;   // p �߳̿���Ŀ,mp ÿ���߳���,ebdΪ���ĩβ����
	int m = p * mp;//mΪ���߳���
	clock_t time1, time2;
	time1 = clock();
	sum_cpu(end);
	time2 = clock();
	printf("Time for CPU run  is: %f seconds \n", (double)(time2 - time1) / CLOCKS_PER_SEC);//����������cpu���㲿��

	time1 = clock();
	double* sum_local_gpu;
	int MM = m * sizeof(double);//��Ҫ���ٵ��ڴ�ռ��С
	double* sum_local_cpu = (double*)malloc(MM);//cpu�����ڴ�
	hipMalloc((void**)&sum_local_gpu, MM);//gpu�����ڴ�

	sum_gpu << <p, mp >> > (end, sum_local_gpu);
	hipDeviceSynchronize();
	hipMemcpy(sum_local_cpu, sum_local_gpu, MM, hipMemcpyDeviceToHost);
	double sum_s = 0.0;
	for (int k = 0; k < m; ++k)//��ÿһ���̵߳���ͽ�������
	{
		sum_s += sum_local_cpu[k];
	}
	printf("sum gpu=%20.16e \n", sum_s);
	time2 = clock();
	printf("Time for GPU run  is: %f seconds \n", (double)(time2 - time1) / CLOCKS_PER_SEC);
	free(sum_local_cpu);
	hipFree(sum_local_gpu);
	return 0;
}
