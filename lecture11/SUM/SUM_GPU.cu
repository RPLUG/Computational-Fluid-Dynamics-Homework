#include "SUM_GPU.cuh"
int main(int argc, char* argv[]){
    hipStream_t* stream;
    stream=(hipStream_t*)malloc(sizeof(hipStream_t));
    memset( stream, 0, sizeof(hipStream_t) );
    hipStreamCreate(&stream[0]);
    LARGE_INTEGER CPU_Start,CPU_End;
    long long until = 2000000000;
    long long CPU_RES;
    
    //Calcu CPU cumsumption
    QueryPerformanceCounter(&CPU_Start);
    CPU_RES = CPU_SUM(until);
    QueryPerformanceCounter(&CPU_End);
    printf("Time used in CPU: %lf, result:%lld\n",cuTime(&CPU_Start,&CPU_End),CPU_RES);
    
    //Calcu sum in GPU;
    LARGE_INTEGER CudaStart,CudaEnd; 
    long long* partial;
    long long* device_ptr;
    int TotalBlock = 1024;
    int Thread = 1024;
    hipHostAlloc(&partial,sizeof(long long)*TotalBlock,hipHostMallocMapped);
    hipHostGetDevicePointer(&device_ptr,partial,0);
    long long* HostRes;
    long long* Resptr;
    hipHostAlloc(&HostRes,sizeof(long long) * 1 , hipHostMallocMapped);
    hipHostGetDevicePointer(&Resptr , HostRes , 0);

    printf("NBlock\tNThread\t\tResult\t\t  Time\n");
    int shared_memory = sizeof(long long) * Thread;
    QueryPerformanceCounter(&CudaStart);

    GPU_SUM <<<TotalBlock , Thread , shared_memory, *stream >>> (until,device_ptr);
    GPU_Reduction <<<1 , TotalBlock, 0 , *stream >>> (device_ptr , Resptr);
    hipDeviceSynchronize();
    QueryPerformanceCounter(&CudaEnd);
    
    printf("%d\t%d\t%lld\t%lf\n",TotalBlock,Thread,HostRes[0],cuTime(&CudaStart,&CudaEnd));

    fflush(stdout);
    hipFree(partial);
    hipFree(HostRes);
    hipFree(device_ptr);
    hipFree(Resptr);
    return 0;
}

