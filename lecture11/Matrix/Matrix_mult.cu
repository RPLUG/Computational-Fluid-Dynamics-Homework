#include "Matrix_mult.cuh"
#include <random>
int main(int argc,char* argv[]){
    int N = 1024;
    int Matrix_size[2] = { N , N };
    int Total_Size = N * N * sizeof(double);
    int N_Block = 1024;
    int N_Thread = 1024;
    //----------------------HOST MALLOC--------------------------

    std::mt19937 generate{std::random_device{}()};
    std::uniform_real_distribution<double> FLY(-10000,10000);
    //A Stored in cols
    double* A=(double*)malloc(sizeof(double)*Total_Size);
    for(int i=0;i<Total_Size;i++){
        A[i]=FLY(generate);
    }
    //B Stored in rows
    double* B=(double*)malloc(sizeof(double)*Total_Size);
    for(int i=0;i<Total_Size;i++){
        B[i]=FLY(generate);
    }
    //C Stored in cols
    double* C=(double*)malloc(sizeof(double)*Total_Size);
    memset(C,0,sizeof(double)*Total_Size);

    //--------------------HOST Check------------------------------
    LARGE_INTEGER CPU_BEG , CPU_END;
    QueryPerformanceCounter( &CPU_BEG );
    Matrix_test( A , B , C , Matrix_size );
    QueryPerformanceCounter( &CPU_END );

    printf("Time Cost in the CPU are %lf, Sum of matrix are %lf\n",cuTime(&CPU_BEG,&CPU_END),Matrix_SUM(C,Total_Size));
    //--------------------CUDA MALLOC-----------------------------
    double* A_CUDA;
    double* B_CUDA;
    double* C_CUDA;
    hipMalloc( &A_CUDA , Total_Size );
    hipMalloc( &B_CUDA , Total_Size );
    hipMalloc( &C_CUDA , Total_Size );
    hipMemcpy( A_CUDA , A , Total_Size , hipMemcpyHostToDevice );
    hipMemcpy( B_CUDA , B , Total_Size , hipMemcpyHostToDevice );
    hipStream_t* stream;
    stream=(hipStream_t*)malloc(sizeof(hipStream_t));
    hipStreamCreate(stream);
    LARGE_INTEGER GPU_START,GPU_END;
    //--------------------CUDA Run-----------------------------
    QueryPerformanceCounter(&GPU_START);
    CUDA_Matrix_Mult<<<N_Block,N_Thread,sizeof(double),*stream>>>(Total_Size,A_CUDA,B_CUDA,C_CUDA,Matrix_size);
    QueryPerformanceCounter(&GPU_END);
    //--------------------CUDA Check---------------------------
    hipMemcpy(C,C_CUDA,Total_Size,hipMemcpyDeviceToHost);
    //----------------------RESULT-----------------------------
    printf("GPU test infomation: \n");
    printf("NBlock\tNthread\t Time\t\tResult\n");
    printf("%d\t%d\t%lf\t%lf\n",N_Block,N_Thread,cuTime(&GPU_START,&GPU_END),Matrix_SUM(C,Total_Size));
    return 0;
}